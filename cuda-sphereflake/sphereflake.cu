#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <memory>
#include <limits>
#include <math.h>
#include <cassert>

using namespace std;

const unsigned int width = 640;
const unsigned int height = (width * 3) / 4;	// Maintain a 4/3 aspect ratio

const char* imageName = "sphereflake.ppm";

const float PI = 3.141592654f;

float DegToRad(float deg)
{
	return deg * ((2.0f * PI) / 360.0f);
}

class Vector3
{
public:
	__host__ __device__ void Set(const float x_, const float y_, const float z_) { x = x_; y = y_; z = z_; }

	__host__ __device__ Vector3& operator+=(const Vector3& b) { x += b.x; y += b.y; z += b.z; return *this; }

	float x;
	float y;
	float z;
};

__host__ __device__ Vector3 MakeVector3(const float x, const float y, const float z) 
{
	Vector3 v;
	v.Set(x, y, z);
	return v;
}

__host__ __device__ float Dot(const Vector3& a, const Vector3& b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ Vector3 operator*(const Vector3& a, const float b)
{
	return MakeVector3(b * a.x, b * a.y, b * a.z);
}

__host__ __device__ Vector3 operator*(const float a, const Vector3& b)
{
	return MakeVector3(a * b.x, a * b.y, a * b.z);
}

__host__ __device__ Vector3 operator+(Vector3 a, const Vector3& b)
{
	a += b;
	return a;
}

__host__ __device__ Vector3 operator-(const Vector3& a, const Vector3& b)
{
	return MakeVector3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ Vector3 operator+(const Vector3& a, const float b)
{
	return MakeVector3(a.x + b, a.y + b, a.z + b);
}

__host__ __device__ Vector3 operator-(const Vector3& a, const float b)
{
	return MakeVector3(a.x - b, a.y - b, a.z - b);
}

__host__ __device__ Vector3 operator+(const float a, const Vector3& b)
{
	return MakeVector3(a + b.x, a + b.y, a + b.z);
}

__host__ __device__ Vector3 Normalize(const Vector3& a)
{
	return rsqrt(Dot(a, a)) * a;
}

__host__ __device__ float Clamp(float x, float a, float b)
{
	return min(max(x, a), b);
}

class Matrix44Affine
{
public:
	enum { NumberOfRows = 4 };
	enum { NumberOfCols = 3 };
	enum { NumberOfElement = 12 };

	void Set(float m0, float m1, float m2, float m3, float m4, float m5,
			 float m6, float m7, float m8, float m9, float m10, float m11)
	{
		xAxis.Set(m0, m1, m2);
		yAxis.Set(m3, m4, m5);
		zAxis.Set(m6, m7, m8);
		wAxis.Set(m9, m10, m11);
	}

	void Set(const Vector3& row0, const Vector3& row1, const Vector3& row2, const Vector3& row3)
	{
		xAxis = row0;
		yAxis = row1;
		zAxis = row2;
		wAxis = row3;
	}

	Vector3 xAxis;
	Vector3 yAxis;
	Vector3 zAxis;
	Vector3 wAxis;
};

Matrix44Affine Matrix44AffineFromScaleTranslation(const Vector3& scale, const Vector3& trans)
{
	const float zero(0.0f);
	Matrix44Affine m;
	m.Set(
		scale.x, zero, zero,
		zero, scale.y, zero,
		zero, zero, scale.z,
		trans.x, trans.y, trans.z
		);
	return m;
}

Matrix44Affine Matrix44AffineFromXRotationAngle(float angle)
{
	float s = sin(angle);
	float c = cos(angle);
	const float zero(0.0f);
	const float one(1.0f);
	Matrix44Affine m;
	m.Set(
		one,    zero,   zero,
		zero,  c,       s,
		zero,  -s,      c,
		zero,  zero,   zero
		);
	return m;
}

Matrix44Affine Matrix44AffineFromYRotationAngle(float angle)
{
	float s = sin(angle);
	float c = cos(angle);
	const float zero(0.0f);
	const float one(1.0f);
	Matrix44Affine m;
	m.Set(
		c,      zero,   -s,
		zero,  one, zero,
		s,      zero,   c,
		zero,  zero,   zero );
	return m;
}

Matrix44Affine Matrix44AffineFromZRotationAngle(float angle)
{
	float s = sin(angle);
	float c = cos(angle);
	const float zero(0.0f);
	const float one(1.0f);
	Matrix44Affine m;
	m.Set(
		c,      s,      zero,
		-s,     c,      zero,
		zero,  zero,   one,
		zero,  zero,   zero);
	return m;
}

float Determinant(const Matrix44Affine& m)
{
	return  
		m.xAxis.x * (
		m.yAxis.y * m.zAxis.z -
		m.yAxis.z * m.zAxis.y 
		) + 
		m.xAxis.y * (
		m.yAxis.z * m.zAxis.x -
		m.yAxis.x * m.zAxis.z
		) + 
		m.xAxis.z * (
		m.yAxis.x * m.zAxis.y -
		m.yAxis.y * m.zAxis.x
		);
}

Matrix44Affine Inverse(const Matrix44Affine& m)
{
	const float determinant = Determinant(m);
	if (determinant == 0.0f) 
	{
		return m;
	}
	else
	{
		float determinantRecip = 1.0f / determinant;
		Matrix44Affine ret;
		ret.Set(
				((m.yAxis.y*(m.zAxis.z)+m.yAxis.z*(-m.zAxis.y))*determinantRecip),
				((m.zAxis.y*(m.xAxis.z)+m.zAxis.z*(-m.xAxis.y))*determinantRecip),
				((m.xAxis.y*m.yAxis.z-m.xAxis.z*m.yAxis.y)*determinantRecip),

				((m.yAxis.z*(m.zAxis.x)+m.yAxis.x*(-m.zAxis.z))*determinantRecip),
				((m.zAxis.z*(m.xAxis.x)+m.zAxis.x*(-m.xAxis.z))*determinantRecip),
				((m.yAxis.x*m.xAxis.z-m.xAxis.x*m.yAxis.z)*determinantRecip),

				((m.yAxis.x*(m.zAxis.y)+m.yAxis.y*(-m.zAxis.x))*determinantRecip),
				((m.zAxis.x*(m.xAxis.y)+m.zAxis.y*(-m.xAxis.x))*determinantRecip),
				((m.xAxis.x*m.yAxis.y-m.yAxis.x*m.xAxis.y)*determinantRecip),

				((m.yAxis.x*(m.wAxis.y*m.zAxis.z-m.zAxis.y*m.wAxis.z)+m.yAxis.y*(m.zAxis.x*m.wAxis.z-m.wAxis.x*m.zAxis.z)+m.yAxis.z*(m.wAxis.x*m.zAxis.y-m.zAxis.x*m.wAxis.y))*determinantRecip),
				((m.zAxis.x*(m.xAxis.z*m.wAxis.y-m.xAxis.y*m.wAxis.z)+m.zAxis.y*(m.xAxis.x*m.wAxis.z-m.wAxis.x*m.xAxis.z)+m.zAxis.z*(m.wAxis.x*m.xAxis.y-m.xAxis.x*m.wAxis.y))*determinantRecip),
				((m.wAxis.x*(m.xAxis.z*m.yAxis.y-m.xAxis.y*m.yAxis.z)+m.wAxis.y*(m.xAxis.x*m.yAxis.z-m.yAxis.x*m.xAxis.z)+m.wAxis.z*(m.yAxis.x*m.xAxis.y-m.xAxis.x*m.yAxis.y))*determinantRecip)
			);
		return ret;
	}
}

__host__ __device__ Vector3 TransformPoint(const Vector3& pt, const Matrix44Affine& matrix)
{
	return MakeVector3(matrix.xAxis.x * pt.x + matrix.yAxis.x * pt.y + matrix.zAxis.x * pt.z + matrix.wAxis.x,
					   matrix.xAxis.y * pt.x + matrix.yAxis.y * pt.y + matrix.zAxis.y * pt.z + matrix.wAxis.y,
					   matrix.xAxis.z * pt.x + matrix.yAxis.z * pt.y + matrix.zAxis.z * pt.z + matrix.wAxis.z);
}

__host__ __device__ Vector3 TransformVector(const Vector3& vec, const Matrix44Affine& matrix)
{
	Vector3 transformedPoint = matrix.xAxis * vec.x;
	transformedPoint += matrix.yAxis * vec.y;
	transformedPoint += matrix.zAxis * vec.z;
	return transformedPoint;
}

Matrix44Affine Mult(const Matrix44Affine& m, const Matrix44Affine& b)
{
	Matrix44Affine ret;
	ret.Set(
		TransformVector(m.xAxis, b),
		TransformVector(m.yAxis, b),
		TransformVector(m.zAxis, b),
		TransformPoint(m.wAxis, b)
		);
	return ret;
}

Matrix44Affine operator*(const Matrix44Affine& a, const Matrix44Affine& b)
{
	return Mult(a, b);
}

class Sphere
{
public:
	__device__ Sphere(const Vector3 &position, const float radius) :m_position(position), m_radius(radius) { }

	__device__ float Intersect(const Vector3& R0, const Vector3& Rd, Vector3& hit) const;

	Vector3 m_position;
	float m_radius;
};

__device__ float Sphere::Intersect(const Vector3& R0, const Vector3& Rd, Vector3& hit) const
{
	// Sphere centered at [Xc, Yc, Zc] with radius r satisfies equation [X-Xc]^2 + [Y-Yc]^2 + [Z-Zc]^2 - r^2 = 0
	// Parametric formulation of ray R has equation R(t) = R0 + t*Rd where R0 is an initial position
	// R0 = [X0, Y0, Z0] and Rd is a direction vector Rd = [Xd, Yd, Zd].
	//
	// Letting Rp = [X0-Xc, Y0-Yc, Z0-Zc] we get the solution
	// t^2(Rd dot Rd) + t(2 * Rd dot Rp) + Rp dot Rp - r^2 = 0
	// which can be solved easily through t = (-b +/- (b^2 - 4*a*c)) / 2*a and can discard if the discriminant is < 0
	
	const Vector3 Rp(R0 - m_position);
	const float radiusSqd(m_radius * m_radius);
	const float A(Dot(Rd,Rd));
	const float B(Dot(Rd,Rp) * 2.0f);
	const float C(Dot(Rp,Rp) - radiusSqd);

	const float discrim((B * B) - (A * C * 4.0f));
	
	if (discrim < 0.0f)
	{
		return FLT_MAX;
	}
	else
	{
		const float sqrtDiscrim = sqrt(discrim);
		const float denominator = 1.0f / (2.0f * A);
		const float t1 = (-B + sqrtDiscrim) * denominator;
		const float t2 = (-B - sqrtDiscrim) * denominator;
		const float tMin = min(t1, t2);
		const float tMax = max(t1, t2);
		
		// Cases:
		// a) tMin, tMax < 0, -> t = FLT_MAX
		// b) tMin < 0 && tMax >= 0 -> t = t2
		// c) tMax >= tMin > 0, -> t = t1
		float t = tMax < 0.0f ? FLT_MAX : (tMin < 0.0f ? tMax : tMin);

		hit = Vector3(R0 + (Rd * t));

		return t;
	}
}

// This bound was determined empirically. (i.e. it looks fine with this bound)
__device__ const float boundingSphereRadius = 8.75f;

class SphereFlake
{
public:
	__device__ SphereFlake(unsigned int levelsOfRecursion, const Vector3 &colorA, const Vector3 &colorB)
		: m_level(levelsOfRecursion),
		  m_centralSphere(MakeVector3(0.0f, 0.0f, 0.0f), 5.0f),
		  m_boundingSphere(MakeVector3(0.0f, 0.0f, 0.0f), boundingSphereRadius)
	{
	}

	__device__ float IntersectImpl(const Vector3& R0, const Vector3& Rd, float bound, Vector3& sphereCenter);
	__device__ float Intersect(const Vector3& R0, const Vector3& Rd, Vector3& hit, Vector3& hitNormal, Vector3& hitColor);

	unsigned int m_level;
	Sphere m_centralSphere;
	Sphere m_boundingSphere;

	const static int NUM_CHILDREN = 7;
	static const float scaleFactor;	// Controls how much smaller each child sphere flake is.
	static const float topChildrenZRotation;
	static const float bottomChildrenZRotation;
	static Matrix44Affine childTransforms[NUM_CHILDREN];
	static Matrix44Affine inverseChildTransforms[NUM_CHILDREN];
};

const float SphereFlake::scaleFactor = 1.0f/3.0f;
const float SphereFlake::topChildrenZRotation = 55.0f;
const float SphereFlake::bottomChildrenZRotation = 110.0f;

Matrix44Affine SphereFlake::childTransforms[SphereFlake::NUM_CHILDREN] =
{
	// The first 3 entries are for the "top" children
	Matrix44Affine(
		Matrix44AffineFromScaleTranslation(MakeVector3(scaleFactor, scaleFactor, scaleFactor), MakeVector3(0.0f, 5.0f+5.0f*scaleFactor, 0.0f)) *
		Matrix44AffineFromZRotationAngle( DegToRad(topChildrenZRotation) ) *
		Matrix44AffineFromYRotationAngle( DegToRad(-30.0f-120.0f*0.0f) )
		),

	Matrix44Affine(
		Matrix44AffineFromScaleTranslation(MakeVector3(scaleFactor, scaleFactor, scaleFactor), MakeVector3(0.0f, 5.0f+5.0f*scaleFactor, 0.0f)) *
		Matrix44AffineFromZRotationAngle( DegToRad(topChildrenZRotation) ) *
		Matrix44AffineFromYRotationAngle( DegToRad(-30.0f-120.0f*1.0f) )
		),

	Matrix44Affine(
		Matrix44AffineFromScaleTranslation(MakeVector3(scaleFactor, scaleFactor, scaleFactor), MakeVector3(0.0f, 5.0f+5.0f*scaleFactor, 0.0f)) *
		Matrix44AffineFromZRotationAngle( DegToRad(topChildrenZRotation) ) *
		Matrix44AffineFromYRotationAngle( DegToRad(-30.0f-120.0f*2.0f) )
		),

	// The last 4 entries are for the "bottom" children
	Matrix44Affine(
		Matrix44AffineFromScaleTranslation(MakeVector3(scaleFactor, scaleFactor, scaleFactor), MakeVector3(0.0f, 5.0f+5.0f*scaleFactor, 0.0f)) *
		Matrix44AffineFromZRotationAngle( DegToRad(bottomChildrenZRotation) ) *
		Matrix44AffineFromYRotationAngle( DegToRad(-45.0f-90.0f*0.0f) )
		),

	Matrix44Affine(
		Matrix44AffineFromScaleTranslation(MakeVector3(scaleFactor, scaleFactor, scaleFactor), MakeVector3(0.0f, 5.0f+5.0f*scaleFactor, 0.0f)) *
		Matrix44AffineFromZRotationAngle( DegToRad(bottomChildrenZRotation) ) *
		Matrix44AffineFromYRotationAngle( DegToRad(-45.0f-90.0f*1.0f) )
		),

	Matrix44Affine(
		Matrix44AffineFromScaleTranslation(MakeVector3(scaleFactor, scaleFactor, scaleFactor), MakeVector3(0.0f, 5.0f+5.0f*scaleFactor, 0.0f)) *
		Matrix44AffineFromZRotationAngle( DegToRad(bottomChildrenZRotation) ) *
		Matrix44AffineFromYRotationAngle( DegToRad(-45.0f-90.0f*2.0f) )
		),

	Matrix44Affine(
		Matrix44AffineFromScaleTranslation(MakeVector3(scaleFactor, scaleFactor, scaleFactor), MakeVector3(0.0f, 5.0f+5.0f*scaleFactor, 0.0f)) *
		Matrix44AffineFromZRotationAngle( DegToRad(bottomChildrenZRotation) ) *
		Matrix44AffineFromYRotationAngle( DegToRad(-45.0f-90.0f*3.0f) )
		),

};

Matrix44Affine SphereFlake::inverseChildTransforms[SphereFlake::NUM_CHILDREN] =
{
	Inverse(childTransforms[0]),
	Inverse(childTransforms[1]),
	Inverse(childTransforms[2]),
	Inverse(childTransforms[3]),
	Inverse(childTransforms[4]),
	Inverse(childTransforms[5]),
	Inverse(childTransforms[6]),
};

__constant__ Matrix44Affine deviceChildTransforms[SphereFlake::NUM_CHILDREN];
__constant__ Matrix44Affine deviceInverseChildTransforms[SphereFlake::NUM_CHILDREN];

class ChildSphereStack
{
public:
	static const int StackSize = 100;

	__device__ ChildSphereStack() : m_top(0) {}

	struct ChildSphereInfo
	{
		int level;
		Vector3 localR0;
		Vector3 localRd;
		Vector3 sphereCenter;
	};

	__device__ void Push(int level, const Vector3& newR0, const Vector3& newRd, const Vector3& newSphereCenter)
	{
		m_stack[m_top].level = level;
		m_stack[m_top].localR0 = newR0;
		m_stack[m_top].localRd = newRd;
		m_stack[m_top].sphereCenter = newSphereCenter;
		++m_top;
	};

	__device__ ChildSphereInfo Pop()
	{
		return m_stack[--m_top];
	}

	__device__ bool Empty()
	{
		return m_top == 0;
	}

	__device__ bool Full()
	{
		return m_top >= StackSize;
	}

	int m_top;
	ChildSphereInfo m_stack[StackSize];
};

__device__ float SphereFlake::IntersectImpl(const Vector3& R0, const Vector3& Rd, const float bound, Vector3& sphereCenter)
{
	float result = bound;
	Vector3 hit(MakeVector3(0.0f, 0.0f, 0.0f));

	ChildSphereStack childSphereStack;

	// Now test all the children.
	childSphereStack.Push(0, R0, Rd, MakeVector3(0.0f, 0.0f, 0.0f));
	
	while (!childSphereStack.Empty())
	{
		ChildSphereStack::ChildSphereInfo csi = childSphereStack.Pop();

		const float boundT = m_boundingSphere.Intersect(csi.localR0, csi.localRd, hit);
		if (boundT > result)
		{
			continue;
		}

		const float t = m_centralSphere.Intersect(csi.localR0, csi.localRd, hit);

		if (t < result)
		{
			result = t;
			sphereCenter = csi.sphereCenter;
		}

		if (csi.level >= m_level)
		{
			continue;
		}

		for (int childIdx = 0; childIdx != NUM_CHILDREN; ++childIdx)
		{
			Matrix44Affine& childTransform = deviceChildTransforms[childIdx];
			Matrix44Affine& inverseChildTransform = deviceInverseChildTransforms[childIdx];

			const Vector3 newR0 = TransformPoint(csi.localR0, inverseChildTransform);
			const Vector3 newRd = TransformVector(csi.localRd, inverseChildTransform);
			const Vector3 newSphereCenter = TransformPoint(csi.sphereCenter, childTransform);

			if (!childSphereStack.Full())
			{
				childSphereStack.Push(csi.level + 1, newR0, newRd, newSphereCenter);
			}
		}
	}

	return result;
}

__device__ float SphereFlake::Intersect(const Vector3& R0, const Vector3& Rd, Vector3& hit, Vector3& hitNormal, Vector3& hitColor)
{
	Vector3 sphereCenter(MakeVector3(0.0f, 0.0f, 0.0f));
	float t = IntersectImpl(R0, Rd, FLT_MAX, sphereCenter);

	hit = Rd * t + R0;

	// Calculate the normal
	hitNormal = Normalize(hit - sphereCenter);
	hitColor = MakeVector3(1.0f, 0.2f, 0.1f);

	return t;
}

__device__ Vector3 ShadeRay(const Vector3 &rayStartPos, const Vector3 &rayDirection, int maxLevelsToRecurse=4)
{
	const Vector3 gLightPos(MakeVector3(100.0f, 100.0f, -100.0f));
	const Vector3 gBackgroundColor(MakeVector3(0.0f, 0.0f, 0.2f));
	const float gAmbientIntensity(0.2f);

	SphereFlake sphereFlake(maxLevelsToRecurse, MakeVector3(1.0f,0.3f, 0.3f), MakeVector3(0.0f, 1.0f, 0.0f));
	Vector3 hitVector(MakeVector3(0.0f, 0.0f, 0.0f));
	Vector3 hitNormal(MakeVector3(0.0f, 0.0f, 1.0f));
	Vector3 hitColor(MakeVector3(0.0f, 0.0f, 0.0f));
	if (sphereFlake.Intersect(rayStartPos, rayDirection, hitVector, hitNormal, hitColor) < FLT_MAX)
	{
			const Vector3 closestHit = hitVector;
			const Vector3 closestHitNormal = hitNormal;
			
			// Calculate light direction
			const Vector3 lightDirection(Normalize(gLightPos - closestHit));

			// Calculate light intensity.
			const float lightIntensity = Clamp(Dot(closestHitNormal, lightDirection), gAmbientIntensity, 1.0f);
			
			// Calculate surface color
			return lightIntensity * hitColor;
	}
	else
	{
		return gBackgroundColor;
	}
}

__device__ Vector3 Raytrace(int x, int y)
{
	const float viewWidth(15.0f);
	const float viewHeight(15.0f);
	const float aspectRatio(4.0f/3.0f);
	float xDelta(aspectRatio * viewWidth / width);
	float yDelta(-viewHeight / height);
	Vector3 rayStartPos(MakeVector3(-viewWidth * 0.5f * aspectRatio + float(x) * xDelta, viewHeight * 0.5f + float(y) * yDelta, -20.0f));
	Vector3 rayDirection(MakeVector3(0.0f, 0.0f, 1.0f));

	return ShadeRay(rayStartPos, rayDirection); 
}

__global__ void sphereflake(uint32_t* devPtr, size_t pitch, size_t width, size_t height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	Vector3 pixColor = Raytrace(x, y);
	uint32_t* pix = devPtr + (pitch / sizeof(uint32_t)) * y + x;
	*pix = ((int(pixColor.x * 255.f) & 0xff) << 16) | ((int(pixColor.y * 255.f) & 0xff) << 8) | ((int(pixColor.z * 255.f) & 0xff) << 0);
}

bool outputPPM(const char *fileName, uint32_t *frameContents)
{
	char PPMHeader[256];
	sprintf(PPMHeader, "P3\n" "%d %d\n" "255\n", width, height);
	char *tempBuffer = new char[width*4*3 + 128];

	printf("Writing PPM file to %s...\n", fileName);
	FILE* file = fopen(fileName, "wb");
	if (!file)
	{
		printf("Error opening file!\n");
		return false;
	}

	fwrite(PPMHeader, strlen(PPMHeader), 1, file);
	
	for (unsigned int y = 0; y < height; ++y)
	{
		char *curPos = tempBuffer;
		for (unsigned int x = 0; x < width; ++x)
		{
			unsigned int col = frameContents[y*width + x];
			unsigned int r = col & 0xff;
			unsigned int g = (col >> 8) & 0xff;
			unsigned int b = (col >> 16) & 0xff;
			curPos += sprintf(curPos, "%d %d %d ", r, g, b);
		}

		sprintf(curPos, "\n");
		fwrite(tempBuffer, strlen(tempBuffer), 1, file);
	}
		   
	fclose(file);
	delete [] tempBuffer;
	return true;
}

#define VERIFY_CUDA_SUCCESS(x) do { hipError_t result = x; assert(hipSuccess == result); } while (false);

int main(int argc, char **argv)
{
	size_t pitch = 0;
	uint32_t* devPtr = nullptr;
	VERIFY_CUDA_SUCCESS(hipMallocPitch(&devPtr, &pitch, width * sizeof(uint32_t), height));

	VERIFY_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(deviceChildTransforms), &SphereFlake::childTransforms, sizeof(deviceChildTransforms)));
	VERIFY_CUDA_SUCCESS(hipMemcpyToSymbol(HIP_SYMBOL(deviceInverseChildTransforms), &SphereFlake::inverseChildTransforms, sizeof(deviceInverseChildTransforms)));

	dim3 dimGrid(width / 16, height / 16);
	dim3 dimBlock(16, 16);
	sphereflake<<<dimGrid, dimBlock>>>(devPtr, pitch, width, height);
	
	VERIFY_CUDA_SUCCESS(hipDeviceSynchronize());

	unique_ptr<uint32_t> output(new uint32_t[width * height]);
	VERIFY_CUDA_SUCCESS(hipMemcpy2D(output.get(), width * sizeof(uint32_t), devPtr, pitch, width * sizeof(uint32_t), height, hipMemcpyDeviceToHost));
	outputPPM(imageName, output.get());

	VERIFY_CUDA_SUCCESS(hipDeviceReset());

	return 0;
}

